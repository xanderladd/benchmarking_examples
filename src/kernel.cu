
#include "Util.h"
#include <stdio.h>
#include <stdlib.h>


int main(int argc, char *argv[])
{
	int num_devices;
    int stim_ind;
    int globalRank;
    int * devGlobalRank;
	hipGetDeviceCount(&num_devices);
	//num_devices = 1;
	for (int i = 0; i < num_devices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//printf("Device Number: %d\n", i);
		//printf("  Device name: %s\n", prop.name);
		//printf("  Memory Clock Rate (KHz): %d\n",		prop.memoryClockRate);
		//printf("  Memory Bus Width (bits): %d\n",	prop.memoryBusWidth);
		//printf("  Peak Memory Bandwidth (GB/s): %f\n\n",		2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
	}
	//RunByModelSerial();// FOR ROY!!!!!!!!!!!!!!! // Run, output VHot and look
    
    if (argc>1){
        stim_ind = char2int(argv[1]); 
        }
        else{
        stim_ind = 0;
        }
     if (argc>2){
         globalRank = char2int(argv[2]); 
         hipMalloc((void**)&devGlobalRank, sizeof(int));
         hipMemcpy(devGlobalRank, &globalRank, sizeof(int), hipMemcpyHostToDevice);
        }
    printf("global rank is %d \n",globalRank );
    printf("NUM DEVICES %d", num_devices);
    printf("got stim num %d\n", stim_ind); 
    printf("using dev %d",stim_ind % num_devices );

	CUDA_RT_CALL(hipSetDevice(stim_ind % num_devices));
	RunByModelP(argc, stim_ind, globalRank );
	return 0;
}

